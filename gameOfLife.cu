


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <stdint.h>
#include <time.h>

#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))

static void HandleError( hipError_t err,const char *file, int line );

__global__ void get_new_status(uint8_t* gridStatus,uint8_t* newStatus, uint32_t gridSize_x, uint32_t gridSize_y, uint32_t iteration_number);
__device__ uint8_t get_neighbour_status(uint32_t xsize, uint32_t ysize,uint32_t xcoord, uint32_t ycoord, uint8_t* gridStatus, uint32_t iteration_number);
__global__ void update_cell_grid(uint8_t* newStatus, uint8_t* gridStatus, uint32_t gridSize_x, uint32_t gridSize_y, uint32_t iteration_number);


//Function that returns an integer array of the status of neighbours when the 
//x,y dimensions of the grid and the current location of cell underconsideration is provided

__device__ uint8_t get_neighbour_status(uint32_t xsize, uint32_t ysize,uint32_t xcoord, uint32_t ycoord, uint8_t* gridStatus, uint32_t iteration_number)
{
	//Sum of the neighbour_status array; Denotes the total alive cells in the neighbours
	uint8_t neighbour_sum = 0;

	//Total size of the cell grid
	uint32_t totalSize = xsize * ysize;


	//Y axis goes from top to bottom
	//Coordinates of neighbours relative to current position
	//The order is right to left and top to bottom excluding the center cell

	int8_t neighbour_x_positions[8] = {-1, 0, 1, -1, 1,-1, 0, 1};
	int8_t neighbour_y_positions[8] = {-1, -1, -1, 0, 0, 1, 1, 1} ;
	
	int neighbour_xcoord, neighbour_ycoord;

	for(int i=0;i<8;i++) {

		neighbour_xcoord = xcoord + (int32_t)neighbour_x_positions[i];
		neighbour_ycoord = ycoord + (int32_t)neighbour_y_positions[i];
	
	//	printf("(nieghbour_xcoord, neighbout_ycoord) :: (%d,%d)\n",neighbour_xcoord, neighbour_ycoord);
	
		//If the neighbour cell is out of bounds assume the cell is dead
		if( (neighbour_xcoord < xsize) && (neighbour_xcoord >= 0) && (neighbour_ycoord < ysize) && (neighbour_ycoord >= 0) ){
			//Getting the status of the neighbour that is still inside the grid
			neighbour_sum = neighbour_sum + gridStatus[(iteration_number-1) * totalSize + neighbour_ycoord * xsize + neighbour_xcoord];
		}
		else {
			neighbour_sum += 0;
		}
	}

	return neighbour_sum;

}

__global__ void get_new_status(uint8_t* gridStatus,uint8_t* newStatus, uint32_t gridSize_x, uint32_t gridSize_y, uint32_t iteration_number)
{
	int x_id = blockIdx.x * blockDim.x + threadIdx.x; 
	int y_id = blockIdx.y * blockDim.y + threadIdx.y;	

	//Total Grid Size
	uint32_t totalSize = gridSize_x * gridSize_y;

	//For each thread/grid box use rules to decide what to do
	//Rules are:
		//Any live cell with fewer than two live neighbors dies, as if by underpopulation.
		//Any live cell with two or three live neighbors lives on to the next generation.
		//Any live cell with more than three live neighbors dies, as if by overpopulation.
		//Any dead cell with exactly three live neighbors becomes a live cell, as if by reproduction.
	
	
	if( (x_id < gridSize_x) && (y_id < gridSize_y) ) {
	
		//Status of the cell in question
		uint8_t cell_status = gridStatus[(iteration_number-1) * totalSize + y_id * gridSize_x + x_id];

		//Getting the total number of alive neighbours
		uint8_t neighbour_status = get_neighbour_status(gridSize_x, gridSize_y, x_id, y_id, gridStatus, iteration_number);

		if((cell_status == 0) && (neighbour_status==3)) {			

		/*	if(neighbour_status == 3){
				newStatus[y_id * gridSize_x + x_id] = 1;
			}
			else{
				newStatus[y_id * gridSize_x + x_id] = 0;
			}
		*/
			newStatus[y_id * gridSize_x + x_id] = 1;

		}
		else if((cell_status == 1) && ((neighbour_status == 2) || (neighbour_status == 3) )) {
		/*	if(neighbour_status  < 2) {				
				newStatus[y_id * gridSize_x + x_id] = 0;
			}
			else if((neighbour_status  == 2) || (neighbour_status == 3)){
				newStatus[y_id * gridSize_x + x_id] = 1;
			}
			if(neighbour_status  > 3){
				newStatus[y_id * gridSize_x + x_id] = 0;
			}
		*/

			newStatus[y_id * gridSize_x + x_id] = 1;			
		}
					
	}
}

//Function that copies oves the new grid values into the original one
__global__ void update_cell_grid(uint8_t* newStatus, uint8_t* gridStatus, uint32_t gridSize_x, uint32_t gridSize_y, uint32_t iteration_number)
{

	int x_id = blockIdx.x * blockDim.x + threadIdx.x; 
	int y_id = blockIdx.y * blockDim.y + threadIdx.y;	
	//Total Grid Size
	uint32_t totalSize = gridSize_x * gridSize_y;
	
	if( (x_id < gridSize_x) && (y_id < gridSize_y) ) {
		gridStatus[iteration_number * totalSize + y_id * gridSize_x + x_id] = newStatus[ y_id * gridSize_x + x_id];
		newStatus[y_id * gridSize_x + x_id] = 0;
	}
}



static void HandleError( hipError_t err,const char *file, int line ) {
    if (err != hipSuccess) {
  		printf( "%s in %s at line %d\n", hipGetErrorString( err ),file, line );
        exit( EXIT_FAILURE );
    }
}


int main()
{

	int number_of_iterations = 1000;
	uint32_t cellGridSize_x = 100;
	uint32_t cellGridSize_y = 100;
	uint32_t totalCellGridSize = cellGridSize_x * cellGridSize_y;

	uint8_t* cell_grids = (uint8_t*)malloc(totalCellGridSize * number_of_iterations * sizeof(uint8_t));
	
	//-----------------------------------------//

	uint8_t* d_cell_grids;
	uint8_t* d_new_positions;
	HANDLE_ERROR(hipMalloc((void**)&d_cell_grids, totalCellGridSize * number_of_iterations * sizeof(uint8_t)));
	HANDLE_ERROR(hipMalloc((void**)&d_new_positions, totalCellGridSize * sizeof(uint8_t)));
	//-----------------------------------------//

	srand(time(0));
	
	//Generating random bits for the first cell grid
	for(uint32_t i = 0; i<totalCellGridSize; i++) {
		cell_grids[i] = rand() % 2;
	}

	//-----------------------------------------//
	//Copying the first grid from host to device
	HANDLE_ERROR(hipMemcpy(d_cell_grids, cell_grids, totalCellGridSize * sizeof(uint8_t),hipMemcpyHostToDevice));
	//-----------------------------------------//
	printf("xsize: %d, ysize: %d\n",(cellGridSize_x + 31)/32,(cellGridSize_y + 31)/32);

	//Setting the block and grid size; Setting the block size to have 32 threads
	dim3 gridSize((cellGridSize_x + 31)/32,cellGridSize_y,1);
	dim3 blockSize(32,1,1);

	for(uint32_t i=1; i<number_of_iterations; i++) {
		get_new_status<<<gridSize,blockSize>>>(d_cell_grids,d_new_positions,cellGridSize_x, cellGridSize_y,i);
		HANDLE_ERROR(hipDeviceSynchronize());	
		update_cell_grid<<<gridSize,blockSize>>>(d_new_positions, d_cell_grids, cellGridSize_x, cellGridSize_y, i);
		HANDLE_ERROR(hipDeviceSynchronize());	
	}
	//-----------------------------------------//
	HANDLE_ERROR(hipMemcpy(cell_grids, d_cell_grids, number_of_iterations * totalCellGridSize * sizeof(uint8_t), hipMemcpyDeviceToHost));
	//-----------------------------------------//
	//Putting the data copied from the Device into a file
	FILE * fp;
	fp = fopen("result.txt","w+");

	for(int h=0; h < number_of_iterations; h++) {
		for(int i=0; i<cellGridSize_y; i++) {
			for(int j=0; j<cellGridSize_x; j++) {
				fprintf(fp,"%d ",cell_grids[h * totalCellGridSize + i * cellGridSize_x + j]);	
			}
			fprintf(fp,"\n");	
		}
	}
	//-----------------------------------------//
	fclose(fp);
	HANDLE_ERROR(hipFree(d_cell_grids));
	HANDLE_ERROR(hipFree(d_new_positions));
	free(cell_grids);
	return 0;
}
